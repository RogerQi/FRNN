#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <tuple>

#include "grid.h"
#include "utils/mink.cuh"
#include "utils/dispatch.cuh"

void SetupGridParams(
    float* bboxes,
    float cell_size,
    GridParams* params) {
  params->grid_min.x = bboxes[0];
  params->grid_max.x = bboxes[1];
  params->grid_min.y = bboxes[2];
  params->grid_max.y = bboxes[3];
  params->grid_min.z = bboxes[4];
  params->grid_max.z = bboxes[5];

  params->grid_size = params->grid_max - params->grid_min;
  params->grid_res.x = (int)(params->grid_size.x / cell_size) + 1;
  params->grid_res.y = (int)(params->grid_size.y / cell_size) + 1;
  params->grid_res.z = (int)(params->grid_size.z / cell_size) + 1;
  params->grid_total = params->grid_res.x * params->grid_res.y * params->grid_res.z;

  params->grid_delta = 1 / cell_size;

  return;
}

void TestSetupGridParamsCUDA(
    at::Tensor bboxes,  // N x 3 x 2 (min, max) at last dimension
    float r) {
  int N = bboxes.size(0);
  // TODO: cell_size determined joint by search radius and bbox_size
  // TODO: cell_size different for different point clouds in the batch
  float cell_size = r;
  // std::cout << "hipMalloc done" << std::endl;
  GridParams* h_params = new GridParams[N];
  for (int i = 0; i < N; ++i) {
    SetupGridParams(
      bboxes.contiguous().data_ptr<float>() + i*6,
      cell_size,
      &h_params[i]
    );
    // std::cout << h_params[i].grid_min.x << ' ' << h_params[i].grid_min.y << ' ' << h_params[i].grid_min.z << std::endl;
    // std::cout << h_params[i].grid_max.x << ' ' << h_params[i].grid_max.y << ' ' << h_params[i].grid_max.z << std::endl;
    // std::cout << h_params[i].grid_size.x << ' ' << h_params[i].grid_size.y << ' ' << h_params[i].grid_size.z << std::endl;
    // std::cout << h_params[i].grid_res.x << ' ' << h_params[i].grid_res.y << ' ' << h_params[i].grid_res.z << std::endl;
    // std::cout << h_params[i].grid_total << ' ' << h_params[i].grid_delta << ' ' << std::endl; 
  }

  // std::cout << "Setup done" << std::endl;

  GridParams* d_params;
  hipMalloc((void**)&d_params, N*sizeof(GridParams));
  hipMemcpy(d_params, h_params, N*sizeof(GridParams), hipMemcpyHostToDevice);

  GridParams* h_d_params = new GridParams[N];
  hipMemcpy(h_d_params, d_params, N*sizeof(GridParams), hipMemcpyDeviceToHost);
  for (int i = 0; i < N; ++i) {
    std::cout << h_d_params[i].grid_min.x << ' ' << h_d_params[i].grid_min.y << ' ' << h_d_params[i].grid_min.z << std::endl;
    std::cout << h_d_params[i].grid_max.x << ' ' << h_d_params[i].grid_max.y << ' ' << h_d_params[i].grid_max.z << std::endl;
    std::cout << h_d_params[i].grid_res.x << ' ' << h_d_params[i].grid_res.y << ' ' << h_d_params[i].grid_res.z << std::endl;
    std::cout << h_d_params[i].grid_total << ' ' << h_d_params[i].grid_delta << ' ' << std::endl; 
    std::cout << h_d_params[i].grid_size.x << ' ' << h_d_params[i].grid_size.y << ' ' << h_d_params[i].grid_size.z << std::endl;
  }
  delete[] h_params;
  delete[] h_d_params;
  hipFree(d_params);
}

template <typename idx_t>
__global__ void InsertPointsKernel(
    const float* __restrict__ points,
    const idx_t* __restrict__ lengths,
    idx_t* grid_cnt, // not sure if we can use __restrict__ here
    idx_t* __restrict__ grid_cell,
    idx_t* __restrict__ grid_idx,
    size_t N,
    size_t P,
    size_t G,
    const GridParams* params) {


  const idx_t chunks_per_cloud = (1 + (P - 1) / blockDim.x);
  const idx_t chunks_to_do = N * chunks_per_cloud;
  for (idx_t chunk=blockIdx.x; chunk < chunks_to_do; chunk += gridDim.x) {
    const idx_t n = chunk / chunks_per_cloud;
    const idx_t start_point = blockDim.x * (chunk % chunks_per_cloud);
    idx_t p = start_point + threadIdx.x;
    if (p >= lengths[n])
      continue;

    float3 grid_min = params[n].grid_min;
    float grid_delta = params[n].grid_delta;
    int3 grid_res = params[n].grid_res;

    int3 gc;
    gc.x = (int) ((points[(n*P+p)*3+0]-grid_min.x) * grid_delta);
    gc.y = (int) ((points[(n*P+p)*3+1]-grid_min.y) * grid_delta);
    gc.z = (int) ((points[(n*P+p)*3+2]-grid_min.z) * grid_delta);

    idx_t gs = (gc.x*grid_res.y + gc.y) * grid_res.z + gc.z;
    grid_cell[n*P+p] = gs;
    // for long, need to convert it to unsigned long long since there is no atomicAdd for long
    // grid_idx[n * P + p] = atomicAdd((unsigned long long*)&grid_cnt[n*grid_total + gs], (unsigned long long)1);
    grid_idx[n*P+p] = atomicAdd(&grid_cnt[n*G + gs], 1);
  } 
}

template<typename idx_t>
void InsertPointsCUDA(
    const at::Tensor points,    // (N, P, 3)
    const at::Tensor lengths,   // (N,)
    at::Tensor grid_cnt,        // (N, G)
    at::Tensor grid_cell,       // (N, P)      
    at::Tensor grid_idx,        // (N, P)
    int G,
    const GridParams* params) { // (N,)
  at::TensorArg points_t{points, "points", 1};
  at::TensorArg lengths_t{lengths, "lengths", 2};
  at::TensorArg grid_cnt_t{grid_cnt, "grid_cnt", 3};
  at::TensorArg grid_cell_t{grid_cell, "grid_cell", 4};
  at::TensorArg grid_idx_t{grid_idx, "grid_idx", 5};

  at::CheckedFrom c = "InsertPointsCUDA";
  at::checkAllSameGPU(c, {points_t, lengths_t, grid_cnt_t, grid_cell_t, grid_idx_t});
  at::checkAllSameType(c, {lengths_t, grid_cnt_t, grid_cell_t, grid_idx_t});

  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int threads = 256;
  int blocks = 256;

  InsertPointsKernel<idx_t><<<blocks, threads, 0, stream>>>(
    points.contiguous().data_ptr<float>(),
    lengths.contiguous().data_ptr<idx_t>(),
    grid_cnt.contiguous().data_ptr<idx_t>(),
    grid_cell.contiguous().data_ptr<idx_t>(),
    grid_idx.contiguous().data_ptr<idx_t>(),
    points.size(0),
    points.size(1),
    G,
    params
  );
  AT_CUDA_CHECK(hipGetLastError());
}

std::tuple<at::Tensor, at::Tensor, at::Tensor> TestInsertPointsCUDA(
    const at::Tensor bboxes,  
    const at::Tensor points,  
    const at::Tensor lengths,
    float r) {
  int N = bboxes.size(0);
  int P = points.size(1);
  float cell_size = r;
  GridParams* h_params = new GridParams[N];
  int max_grid_total = 0;
  for (size_t i = 0; i < N; ++i) {
    SetupGridParams(
      bboxes.contiguous().data_ptr<float>() + i*6,
      cell_size,
      &h_params[i]
    );
    max_grid_total = std::max(max_grid_total, h_params[i].grid_total);
  }

  GridParams* d_params;
  hipMalloc((void**)&d_params, N*sizeof(GridParams));
  hipMemcpy(d_params, h_params, N*sizeof(GridParams), hipMemcpyHostToDevice);

  auto long_dtype = lengths.options().dtype(at::kLong);
  auto int_dtype = lengths.options().dtype(at::kInt);

  auto dtype = long_dtype;
  dtype = int_dtype;

  auto grid_cnt = at::zeros({N, max_grid_total}, dtype);
  auto grid_cell = at::full({N, P}, -1, dtype); 
  auto grid_idx = at::full({N, P}, -1, dtype);

  InsertPointsCUDA<int>(
    points,
    lengths,
    grid_cnt,
    grid_cell,
    grid_idx,
    max_grid_total,
    d_params
  );

  delete[] h_params;
  hipFree(d_params);
  return std::make_tuple(grid_cnt, grid_cell, grid_idx);
}

template<int K>
__global__ void FindNbrsKernel(
    const float* __restrict__ points1,       
    const float* __restrict__ points2,       
    const int* __restrict__ lengths1,        
    const int* __restrict__ lengths2,
    const int* __restrict__ grid_off,
    const int* __restrict__ sorted_point_idx,
    float* __restrict__ dists,               
    int* __restrict__ idxs,                  
    const int N,
    const int P1,
    const int P2,
    const int G,
    const GridParams* params,                   // (N,)
    float r) {
  float min_dists[K];
  int min_idxs[K];
  float3 diff;
  float sqdist;
  
  const int chunks_per_cloud = (1 + (P1 - 1) / blockDim.x);
  const int chunks_to_do = N * chunks_per_cloud;
  for (int chunk = blockIdx.x; chunk < chunks_to_do; chunk += gridDim.x) {
    const int n = chunk / chunks_per_cloud;
    const int start_point = blockDim.x * (chunk % chunks_per_cloud);
    int p1 = start_point + threadIdx.x;
    if (p1 >= lengths1[n])
      continue;
    float3 cur_point;
    cur_point.x = points1[n*P1*3 + p1*3];
    cur_point.y = points1[n*P1*3 + p1*3 + 1];
    cur_point.z = points1[n*P1*3 + p1*3 + 2];
    int3 res = params[n].grid_res;
    float3 grid_min = params[n].grid_min;
    float grid_delta = params[n].grid_delta;
    float r2 = r*r;

    int3  min_gc, max_gc;
    // gc.x = (int) ((cur_point.x-grid_min.x) * grid_delta);
    // gc.y = (int) ((cur_point.y-grid_min.y) * grid_delta);
    // gc.z = (int) ((cur_point.z-grid_min.z) * grid_delta);
    min_gc.x = (int) ((cur_point.x-grid_min.x-r) * grid_delta);
    min_gc.y = (int) ((cur_point.y-grid_min.y-r) * grid_delta);
    min_gc.z = (int) ((cur_point.z-grid_min.z-r) * grid_delta);
    max_gc.x = (int) ((cur_point.x-grid_min.x+r) * grid_delta);
    max_gc.y = (int) ((cur_point.y-grid_min.y+r) * grid_delta);
    max_gc.z = (int) ((cur_point.z-grid_min.z+r) * grid_delta);
    for (int x=std::max(min_gc.x, 0); x<std::min(max_gc.x, res.x-1); ++x) {
      for (int y=std::max(min_gc.y, 0); y<std::min(max_gc.y, res.y-1); ++y) {
        for (int z=std::max(min_gc.z, 0); z<std::min(max_gc.z, res.z-1); ++z) {
          int cell_idx = (x*res.y + y)*res.z + z;
          int p2_start = grid_off[n*G + cell_idx];
          int p2_end;
          if (cell_idx+1 == params[n].grid_total) {
            p2_end = lengths2[n];
          }
          else {
            p2_end = grid_off[n*G+cell_idx+1]; 
          }
          MinK<float, int> mink(min_dists, min_idxs, K);
          for (int p2=p2_start; p2<p2_end; ++p2) {
            diff.x = points2[n*P2*3 + p2*3] - cur_point.x;
            diff.y = points2[n*P2*3 + p2*3 + 1] - cur_point.y;
            diff.z = points2[n*P2*3 + p2*3 + 2] - cur_point.z;
            sqdist = diff.x*diff.x + diff.y*diff.y + diff.z*diff.z;
            if (sqdist <= r2) {
              mink.add(sqdist, sorted_point_idx[p2]);
            }
          }
          mink.sort();
          for (int k=0; k < mink.size(); ++k) {
            idxs[n*P1*K + p1*K + k] = min_idxs[k];
            dists[n*P1*K + p1*K + k] = min_dists[k];
          }
        }
      }
    }
  }
}
/*
// used for DispatchKernel1D
template<int K>
struct FindNbrsKernelFunctor {
  static void run(
    size_t blocks,
    size_t threads,
    const float* __restrict__ points1,          // (N, P1, 3)
    const float* __restrict__ points2,          // (N, P2, 3)
    const int* __restrict__ lengths1,           // (N,)
    const int* __restrict__ lengths2,           // (N,)
    const int* __restirct__ grid_off,           // (N, G)
    const int* __restrict__ sorted_point_idx,   // (N, P)
    float* __restrict__ dists,                  // (N, P1, K)
    int* __restrict__ idxs,                     // (N, P1, K)
    int N,
    int P1,
    int P2,
    int G,
    const GridParams* params,                   // (N,)
    float r) {
      hipStream_t stream = at::cuda::getCurrentCUDAStream();
      FindNbrsKernel<<blocks, threads, 0, stream>>>(
        points1, points2, lengths1, lengths2, grid_off, sorted_point_idx,
        dists, idxs, N, P1, P2, G, params, r);
    }
};

constexpr int MIN_K = 1;
constexpr int MAX_K = 32;
*/
std::tuple<at::Tensor, at::Tensor> FindNbrsCUDA(
    const at::Tensor points1,
    const at::Tensor points2,
    const at::Tensor lengths1,
    const at::Tensor lengths2,
    const at::Tensor grid_off,
    const at::Tensor sorted_point_idx,
    const GridParams* params,
    int K,
    float r) {
  at::TensorArg points1_t{points1, "points1", 1};
  at::TensorArg points2_t{points2, "points2", 2};
  at::TensorArg lengths1_t{lengths1, "lengths1", 3};
  at::TensorArg lengths2_t{lengths2, "lengths2", 4};
  at::TensorArg grid_off_t{grid_off, "grid_off", 5};
  at::TensorArg sorted_point_idx_t{sorted_point_idx, "sorted_point_idx", 6};

  at::CheckedFrom c = "FindNbrsCUDA";
  at::checkAllSameGPU(c, {points1_t, points2_t, lengths1_t, lengths2_t, grid_off_t, sorted_point_idx_t});
  at::checkAllSameType(c, {points1_t, points2_t});
  at::checkAllSameType(c, {lengths1_t, lengths2_t});
  at::checkAllSameType(c, {grid_off_t, sorted_point_idx_t});
  at::cuda::CUDAGuard device_guard(points1.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int N = points1.size(0);
  const int P1 = points1.size(1);
  const int P2 = points2.size(1);
  const int G = grid_off.size(1);
  
  auto int_dtype = lengths1.options().dtype(at::kInt);
  auto idxs = at::full({N, P1, K}, -1, int_dtype);
  auto dists = at::zeros({N, P1, K}, points1.options());

  const size_t threads = 256;
  const size_t blocks = 256;

  //DispatchKernel1D<FindNbrsKernelFunctor, MIN_K, MAX_K>( 
  //  K,
  //  blocks,
  //  threads,
  
  FindNbrsKernel<5><<<blocks, threads, 0, stream>>>(
    points1.contiguous().data_ptr<float>(),
    points2.contiguous().data_ptr<float>(),
    lengths1.contiguous().data_ptr<int>(),
    lengths2.contiguous().data_ptr<int>(),
    grid_off.contiguous().data_ptr<int>(),
    sorted_point_idx.contiguous().data_ptr<int>(),
    dists.data_ptr<float>(),
    idxs.data_ptr<int>(),
    N,
    P1,
    P2,
    G,
    params,
    r
  );

  return std::make_tuple(idxs, dists);
}
