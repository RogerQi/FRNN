#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <float.h>
#include <iostream>
#include <tuple>

#include "grid.h"
#include "utils/prefix_sum.cuh"
#include "utils/counting_sort.cuh"
#include "utils/mink.cuh"



void SetupGridParamsCUDA(
    float* points_max,
    float* points_min,
    float cell_size,
    GridParams& params) {
    // no documentation for at::max so I just do it myself
    std::cout << "setup grid params" << std::endl;

    params.gridMin.x = points_min[0];
    params.gridMin.y = points_min[1];
    params.gridMin.z = points_min[2];
    params.gridMax.x = points_max[0];
    params.gridMax.y = points_max[1];
    params.gridMax.z = points_max[2];

    std::cout << params.gridMin.x << ' ' << params.gridMin.y << ' ' << params.gridMin.z << std::endl;
    std::cout << params.gridMax.x << ' ' << params.gridMax.y << ' ' << params.gridMax.z << std::endl;
    
    params.gridSize = params.gridMax - params.gridMin;
    params.gridCellSize = cell_size;
    params.gridRes.x = (int)(params.gridSize.x / cell_size) + 1;
    params.gridRes.y = (int)(params.gridSize.y / cell_size) + 1;
    params.gridRes.z = (int)(params.gridSize.z / cell_size) + 1;
    params.gridDelta = 1 / cell_size;
    std::cout << "grid delta done" << std::endl;

    params.gridTotal = params.gridRes.x * params.gridRes.y * params.gridRes.z;
    params.gridSrch = 2;

    std::cout << "grid srch done" << std::endl;
}

__global__ void InsertPointsCUDAKernel(
        const float* __restrict__ Points,
        // int* __restrict__ Grid,
        int* GridCnt,       // not sure if we can use __restrict__ here cause its value would be read
        int* __restrict__ GridCell,
        int* __restrict__ GridIdx,
        int num_points,
        const GridParams* params) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_points) return;

    register float3 gridMin = params->gridMin;
    register float gridDelta = params->gridDelta;
    register int3 gridRes = params->gridRes;
    // printf("params set done");
    // register float3 gridMin;
    // gridMin.x = 0.; gridMin.y = 0., gridMin.z = 0.;
    // register float gridDelta = 10.;
    // register int3 gridRes;
    // gridRes.x = 10; gridRes.y = 10; gridRes.z = 10;
    
    register int gs;
    register int3 gc;

    gc.x = (int) ((Points[i*3+0]-gridMin.x) * gridDelta);
    gc.y = (int) ((Points[i*3+1]-gridMin.y) * gridDelta);
    gc.z = (int) ((Points[i*3+2]-gridMin.z) * gridDelta);

    gs = (gc.x*gridRes.y + gc.y) * gridRes.z + gc.z;
    GridCell[i] = gs;
    GridIdx[i] = atomicAdd(&GridCnt[gs], 1);
}

void InsertPointsCUDA(
        const at::Tensor Points,
        at::Tensor Grid,
        at::Tensor GridCnt,
        at::Tensor GridCell,
        at::Tensor GridIdx,
        const GridParams* params) {
    at::TensorArg Points_t{Points, "Points", 1};
    at::TensorArg Grid_t{Grid, "Grid", 2};
    at::TensorArg GridCnt_t{GridCnt, "GridCnt", 3};
    at::TensorArg GridCell_t{GridCell, "GridCell", 4};
    at::TensorArg GridIdx_t{GridIdx, "GridIdx", 5};

    at::CheckedFrom c = "InsertPointsCUDA";
    at::checkAllSameGPU(c, {Points_t, Grid_t, GridCnt_t, GridCell_t, GridIdx_t});
    at::checkAllSameType(c, {Grid_t, GridCnt_t, GridCell_t, GridIdx_t});

    at::cuda::CUDAGuard device_guard(Points.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    int threadsPerBlock = 192;  // Not sure about this value
    int numBlocks = (int)std::ceil((float)Points.size(0) / threadsPerBlock);

    InsertPointsCUDAKernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
        Points.contiguous().data_ptr<float>(),
        GridCnt.contiguous().data_ptr<int>(),
        GridCell.contiguous().data_ptr<int>(),
        GridIdx.contiguous().data_ptr<int>(),
        Points.size(0),
        params
    );
    hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr,  "CUDA ERROR: InsertPointsCUDA: %s\n", hipGetErrorString(error) );
	}  
	hipDeviceSynchronize();
}

std::tuple<at::Tensor, at::Tensor> TestGridCUDA(
        const at::Tensor Points,
        const at::Tensor bbox_max,
        const at::Tensor bbox_min,
        int K,
        float r) {
    std::cout << "enter TestGrid" << std::endl;
    float r2 = r * r;
    float cell_size = r;
    GridParams params;
    int num_points = Points.size(0);
    SetupGridParamsCUDA(
        bbox_max.contiguous().data_ptr<float>(),
        bbox_min.contiguous().data_ptr<float>(),
        cell_size, params);
    GridParams* d_params;
    // std::cout << "d_params start" << std::endl;
    hipMalloc((void**)&d_params, sizeof(GridParams));
    // std::cout << "d_params allocated" << std::endl;
    hipMemcpy(d_params, &params, sizeof(GridParams), hipMemcpyHostToDevice);
    // std::cout << "d_params copied" << std::endl;
    // std::cout << params.gridMax.x << ' ' << params.gridMax.y << ' ' << params.gridMax.z << std::endl;
    // std::cout << "grid params setup done" << std::endl;

    auto int_dtype = Points.options().dtype(at::kInt);
    
    // not used right now
    at::Tensor Grid = at::full({params.gridTotal}, -1, int_dtype);
    // cell -> #points in this cell
    at::Tensor GridCnt = at::zeros({params.gridTotal}, int_dtype);
    // Point -> cell idx
    at::Tensor GridCell = at::full({num_points}, -1, int_dtype);
    // Point -> next point idx in the same cell
    // at::Tensor GridNext = at::full({num_points}, -1, int_dtype);
    // Point -> idx in its cell
    at::Tensor GridIdx = at::full({num_points}, -1, int_dtype);
    
    // new Points and GridCell after sorting
    at::Tensor SortedGridCell = at::zeros({num_points}, GridCell.options());
    at::Tensor SortedPoints = at::zeros({num_points, 3}, Points.options());
    at::Tensor SortedIdx = at::zeros({num_points}, GridCell.options());

    InsertPointsCUDA(Points, Grid, GridCnt, GridCell, GridIdx, d_params);
    std::cout << "points inserted" << std::endl;
    // return std::make_tuple(GridCnt, GridCell);

    at::Tensor GridOff = PrefixSum(GridCnt);
    CountingSortFullCUDA (
        GridCell, 
        GridIdx, 
        GridOff, 
        Points,
        SortedGridCell,
        SortedPoints,
        SortedIdx
    );

    return FindNbrsGridCUDA(SortedPoints, GridOff, SortedGridCell, SortedIdx, d_params, K, r2);
}

at::Tensor PrefixSum(at::Tensor GridCnt) {
    int num_grids = GridCnt.size(0);
    at::Tensor GridOff = at::zeros({num_grids}, GridCnt.options());
    preallocBlockSumsInt(num_grids);
    prescanArrayRecursiveInt(
        GridOff.contiguous().data_ptr<int>(),
        GridCnt.contiguous().data_ptr<int>(),
        num_grids,
        0
    );
    hipDeviceSynchronize();
    deallocBlockSumsInt();
    return GridOff;
}

// fix K to be 5 for now
// template later
__global__ void FindNbrsGridKernel(
    const float* __restrict__ Points, 
    const int* __restrict__ GridOff,
    const int* __restrict__ GridCell,
    const int* __restrict__ SortedIdx,
    float* __restrict__ dists,
    long* __restrict__ idxs,
    const GridParams* params,
    int num_points,
    float r2) {

    const int K = 5;

    int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= num_points) return;
    float3 dist;
    float dsq;

    register float px = Points[i*3], py = Points[i*3+1], pz = Points[i*3+2];
    register int res_x = params->gridRes.x, res_y = params->gridRes.y, res_z = params->gridRes.z;
    register int grid_srch = params->gridSrch;
    register int grid_total = params->gridTotal;
    int grid_idx = GridCell[i];
    // gs = gc.x*params.gridRes.y + gc.y)*params.gridRes.z + gc.z
    int cz = grid_idx % res_z;
    int cy = (grid_idx / res_z) % res_y;
    int cx = (grid_idx / res_z) / res_y;
    // printf("%f %f %f %d %d %d\n", px, py, pz, cx, cy, cz);
    int startx = std::max(0, cx-grid_srch), endx = std::min(cx+grid_srch, res_x-1);
    int starty = std::max(0, cy-grid_srch), endy = std::min(cy+grid_srch, res_y-1);
    int startz = std::max(0, cz-grid_srch), endz = std::min(cz+grid_srch, res_z-1);

    int original_i = SortedIdx[i];
    if (original_i == 1) {
        printf("%d %d %d %d %d %d %d\n", i, startx, endx, starty, endy, startz, endz);
    }

    float min_dists[5];
    int min_idxs[5];
    MinK<float, int> mink(min_dists, min_idxs, K);
    for (int x=startx; x<=endx; ++x) {
        for (int y=starty; y<=endy; ++y) {
            for (int z=startz; z<=endz; ++z) {
                int cur = (x*res_y + y)*res_z + z;
                // int p_start = ((cur-1) >= 0 ? GridOff[cur-1] : 0);
                // int p_end = GridOff[cur];
                int p_start = GridOff[cur];
                int p_end = cur+1 == grid_total ? num_points : GridOff[cur+1];

                // if (original_i == 1) {
                //     printf("%d %d %d %d %d %d\n", x, y, z, cur, p_start, p_end);
                // }

                for (int p=p_start; p < p_end; ++p) {
                    dist.x = Points[p*3] - px;
                    dist.y = Points[p*3+1] - py;
                    dist.z = Points[p*3+2] - pz;
                    dsq = dist.x*dist.x + dist.y*dist.y + dist.z*dist.z;
                    if (dsq <= r2) {
                        // printf("%f %f %f %f %f %f %d %d %f\n", px, py, pz, Points[p*3], Points[p*3+1], Points[p*3+2], i, p, dsq);
                        mink.add(dsq, SortedIdx[p]);
                    }
                }
                mink.sort();
                for (int k=0; k < mink.size(); ++k) {
                    idxs[original_i*K+k] = min_idxs[k];
                    dists[original_i*K+k] = min_dists[k];
                }
            }
        }
    }
}

std::tuple<at::Tensor, at::Tensor> FindNbrsGridCUDA(
    at::Tensor Points,
    at::Tensor GridOff,
    at::Tensor GridCell,
    at::Tensor SortedIdx,
    const GridParams* params,
    int K,
    float r2) 
{
    int threadsPerBlock = 192;  // Not sure about this value
    int numBlocks = (int)std::ceil((float)Points.size(0) / threadsPerBlock);
    int num_points = Points.size(0);

    at::TensorArg Points_t{Points, "Points", 1};
    at::TensorArg GridOff_t{GridOff, "GridOff", 2};
    at::TensorArg GridCell_t{GridCell, "GridCell", 3};

    at::CheckedFrom c = "FindNbrsGridCUDA";
    at::checkAllSameGPU(c, {Points_t, GridOff_t, GridCell_t});
    at::checkAllSameType(c, {GridOff_t, GridCell_t});

    at::cuda::CUDAGuard device_guard(Points.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();



    auto long_opts = Points.options().dtype(torch::kInt64);
    torch::Tensor idxs = torch::full({num_points, K}, -1, long_opts);
    torch::Tensor dists = torch::full({num_points, K}, -1, Points.options());
    
    FindNbrsGridKernel <<< numBlocks, threadsPerBlock, 0, stream >>> (
        Points.contiguous().data_ptr<float>(),
        GridOff.contiguous().data_ptr<int>(),
        GridCell.contiguous().data_ptr<int>(),
        SortedIdx.contiguous().data_ptr<int>(),
        dists.contiguous().data_ptr<float>(),
        idxs.contiguous().data_ptr<long>(),
        params,
        num_points,
        r2
    );
    return std::make_tuple(idxs, dists);
}

/*
__global__ void FindNbrsGridNoSortingKernel(
    const float* __restrict__ Points, 
    const int* __restrict__ Grid,
    const int* __restrict__ GridNext,
    const int* __restrict__ GridCell,
    float* __restrict__ dists,
    long* __restrict__ idxs,
    const GridParams* params,
    int num_points,
    float r2) {

    const int K = 5;

    int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= num_points) return;

    float3 dist;
    float dsq;

    register float px = Points[i*3], py = Points[i*3+1], pz = Points[i*3+2];
    register int res_x = params->gridRes.x, res_y = params->gridRes.y, res_z = params->gridRes.z;
    register int grid_srch = params->gridSrch;
    int grid_idx = GridCell[i];
    // gs = gc.x*params.gridRes.y + gc.y)*params.gridRes.z + gc.z
    int cz = grid_idx % res_z;
    int cy = (grid_idx / res_z) % res_y;
    int cx = (grid_idx / res_z) / res_y;
    // printf("%f %f %f %d %d %d\n", px, py, pz, cx, cy, cz);
    int startx = std::max(0, cx-grid_srch), endx = std::min(cx+grid_srch, res_x-1);
    int starty = std::max(0, cy-grid_srch), endy = std::min(cy+grid_srch, res_y-1);
    int startz = std::max(0, cz-grid_srch), endz = std::min(cz+grid_srch, res_z-1);

    float min_dists[5];
    int min_idxs[5];
    MinK<float, int> mink(min_dists, min_idxs, K);
    for (int x=startx; x<=endx; ++x) {
        for (int y=starty; y<=endy; ++y) {
            for (int z=startz; z<=endz; ++z) {
                int grid_idx = (x*res_y + y)*res_z + z
                int cur = Grid[grid_idx] 
                while (cur != -1) {
                    if (cur != i || true) {
                        dist.x = Points[cur*3] - px;
                        dist.y = Points[cur*3+1] - py;
                        dist.z = Points[cur*3+2] - pz;
                        dsq = dist.x*dist.x + dist.y*dist.y + dist.z*dist.z;
                        if (dsp <= r2) {
                            mink.add(dsq, cur);
                        }
                    }
                    cur = GridNext[cur];
                }
                mink.sort();
                for (int k = 0; k < mink.size(); ++k) {
                    idxs[i*K+k] = min_idxs[k];
                    dists[i*K+k] = min_dists[k];
                }
            }
        }
    }
}


std::tuple<at::Tensor, at::Tensor> FindNbrsGridNoSortingCUDA(
    at::Tensor Points,
    at::Tensor Grid,
    at::Tensor GridCell,
    at::Tensor GridNext,
    const GridParams* params,
    int K,
    float r2) 
{
    int threadsPerBlock = 192;  // Not sure about this value
    int numBlocks = (int)std::ceil((float)Points.size(0) / threadsPerBlock);
    int num_points = Points.size(0);

    at::TensorArg Points_t{Points, "Points", 1};
    at::TensorArg Grid_t{Grid, "GridOff", 2};
    at::TensorArg GridCell_t{GridCell, "GridCell", 3};
    at::TensorArg GridNext_t{GridNext, "GridCell", 3};

    at::CheckedFrom c = "FindNbrsGridNoSortingCUDA";
    at::checkAllSameGPU(c, {Points_t, Grid_t, GridCell_t, GridNext_t});
    at::checkAllSameType(c, {Grid_t, GridCell_t, GridNext_t});

    at::cuda::CUDAGuard device_guard(Points.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    auto long_opts = Points.options().dtype(torch::kInt64);
    torch::Tensor idxs = torch::full({num_points, K}, -1, long_opts);
    torch::Tensor dists = torch::full({num_points, K}, -1, Points.options());
    
    FindNbrsGridNoSortingKernel <<< numBlocks, threadsPerBlock, 0, stream >>> (
        Points.contiguous().data_ptr<float>(),
        Grid.contiguous().data_ptr<int>(),
        GridCell.contiguous().data_ptr<int>(),
        GridNext.contiguous().data_ptr<int>(),
        dists.contiguous().data_ptr<float>(),
        idxs.contiguous().data_ptr<long>(),
        params,
        num_points,
        r2
    );
    return std::make_tuple(idxs, dists);
}


__global__ void InsertPointsNoSortingKernel(
        const float* __restrict__ Points,
        int* Grid,
        int* GridCnt,       // not sure if we can use __restrict__ here cause its value would be read
        int* __restrict__ GridCell,
        int* __restrict__ GridNext,
        int num_points,
        const GridParams* params) {

    int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i >= num_points) return;

    register float3 gridMin = params->gridMin;
    register float gridDelta = params->gridDelta;
    register int3 gridRes = params->gridRes;
    
    register int gs;
    register int3 gc;

    gc.x = (int) ((Points[i*3+0]-gridMin.x) * gridDelta);
    gc.y = (int) ((Points[i*3+1]-gridMin.y) * gridDelta);
    gc.z = (int) ((Points[i*3+2]-gridMin.z) * gridDelta);

    // Problem here: I don't know how to parallel here
    gs = (gc.x*gridRes.y + gc.y) * gridRes.z + gc.z;
    GridCell[i] = gs;
    GridNext[i] = Grid[gs];
    Grid[gs] = i;
    atomicAdd(&GridCnt[gs], 1);
}

std::tuple<at::Tensor, at::Tensor> TestGridNoSortingCUDA(
        const at::Tensor Points,
        const at::Tensor bbox_max,
        const at::Tensor bbox_min,
        int K,
        float r) {
    std::cout << "enter TestGridNoSortingCUDA" << std::endl;
    float r2 = r * r;
    float cell_size = r;
    GridParams params;
    int num_points = Points.size(0);
    SetupGridParamsCUDA(
        bbox_max.contiguous().data_ptr<float>(),
        bbox_min.contiguous().data_ptr<float>(),
        cell_size, 
        params);
    GridParams* d_params;
    hipMalloc((void**)&d_params, sizeof(GridParams));
    hipMemcpy(d_params, &params, sizeof(GridParams), hipMemcpyHostToDevice);

    auto int_dtype = Points.options().dtype(at::kInt);
    
    // cell -> first point's idx in this cell
    at::Tensor Grid = at::full({params.gridTotal}, -1, int_dtype);
    // cell -> #points in this cell
    at::Tensor GridCnt = at::zeros({params.gridTotal}, int_dtype);
    // Point -> cell idx
    at::Tensor GridCell = at::full({num_points, 3}, -1, int_dtype);
    // Point -> next point idx in the same cell
    at::Tensor GridNext = at::full({num_points}, -1, int_dtype);
    // Point -> idx in its cell
    // at::Tensor GridIdx = at::full({num_points}, -1, int_dtype);
    
    // new Points and GridCell after sorting
    // at::Tensor SortedGridCell = at::zeros({num_points}, GridCell.options());
    // at::Tensor SortedPoints = at::zeros({num_points, 3}, Points.options());

    InsertPointsCUDA(Points, Grid, GridCnt, GridCell, GridIdx, d_params);
    std::cout << "points inserted" << std::endl;

    at::Tensor GridOff = PrefixSum(GridCnt);
    CountingSortFullCUDA (
        GridCell, 
        GridIdx, 
        GridOff, 
        Points,
        SortedGridCell,
        SortedPoints
    );

    return FindNbrsGridCUDA(SortedPoints, GridOff, SortedGridCell, d_params, K, r2);
}
*/
