#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <float.h>
#include <iostream>
#include <tuple>

#include "grid.h"
#include "utils/prefix_sum.cuh"
#include "utils/mink.cuh"



void SetupGridParamsCUDA(
    float* points_max,
    float* points_min,
    float cell_size,
    GridParams& params) {
    // no documentation for at::max so I just do it myself
    std::cout << "setup grid params" << std::endl;

    params.gridMin.x = points_min[0];
    params.gridMin.y = points_min[1];
    params.gridMin.z = points_min[2];
    params.gridMax.x = points_max[0];
    params.gridMax.y = points_max[1];
    params.gridMax.z = points_max[2];
    
    params.gridSize = params.gridMax - params.gridMin;
    params.gridCellSize = cell_size;
    params.gridRes.x = (int)(params.gridSize.x / cell_size) + 1;
    params.gridRes.y = (int)(params.gridSize.y / cell_size) + 1;
    params.gridRes.z = (int)(params.gridSize.z / cell_size) + 1;
    params.gridDelta = 1 / cell_size;
    std::cout << "grid delta done" << std::endl;

    params.gridTotal = params.gridRes.x * params.gridRes.y * params.gridRes.z;
    params.gridSrch = 1;

    std::cout << "grid srch done" << std::endl;
}

__global__ void InsertPointsCUDAKernel(
        const float* __restrict__ Points,
        // int* __restrict__ Grid,
        int* GridCnt,       // not sure if we can use __restrict__ here cause its value would be read
        int* __restrict__ GridCell,
        int* __restrict__ GridIdx,
        int num_points,
        const GridParams* params) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_points) return;

    register float3 gridMin = params->gridMin;
    register float gridDelta = params->gridDelta;
    register int3 gridRes = params->gridRes;
    // printf("params set done");
    // register float3 gridMin;
    // gridMin.x = 0.; gridMin.y = 0., gridMin.z = 0.;
    // register float gridDelta = 10.;
    // register int3 gridRes;
    // gridRes.x = 10; gridRes.y = 10; gridRes.z = 10;
    
    register int gs;
    register int3 gc;

    gc.x = (int) ((Points[i*3+0]-gridMin.x) * gridDelta);
    gc.y = (int) ((Points[i*3+1]-gridMin.y) * gridDelta);
    gc.z = (int) ((Points[i*3+2]-gridMin.z) * gridDelta);

    gs = (gc.x*gridRes.y + gc.y) * gridRes.z + gc.z;
    GridCell[i] = gs;
    GridIdx[i] = atomicAdd(&GridCnt[gs], 1);
}

void InsertPointsCUDA(
        const at::Tensor Points,
        at::Tensor Grid,
        at::Tensor GridCnt,
        at::Tensor GridCell,
        at::Tensor GridIdx,
        const GridParams* params) {
    at::TensorArg Points_t{Points, "Points", 1};
    at::TensorArg Grid_t{Grid, "Grid", 2};
    at::TensorArg GridCnt_t{GridCnt, "GridCnt", 3};
    at::TensorArg GridCell_t{GridCell, "GridCell", 4};
    at::TensorArg GridIdx_t{GridIdx, "GridIdx", 5};

    at::CheckedFrom c = "InsertPointsCUDA";
    at::checkAllSameGPU(c, {Points_t, Grid_t, GridCnt_t, GridCell_t, GridIdx_t});
    at::checkAllSameType(c, {Grid_t, GridCnt_t, GridCell_t, GridIdx_t});

    at::cuda::CUDAGuard device_guard(Points.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    int threadsPerBlock = 192;  // Not sure about this value
    int numBlocks = (int)std::ceil((float)Points.size(0) / threadsPerBlock);

    InsertPointsCUDAKernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
        Points.contiguous().data_ptr<float>(),
        GridCnt.contiguous().data_ptr<int>(),
        GridCell.contiguous().data_ptr<int>(),
        GridIdx.contiguous().data_ptr<int>(),
        Points.size(0),
        params
    );
    hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr,  "CUDA ERROR: InsertPointsCUDA: %s\n", hipGetErrorString(error) );
	}  
	hipDeviceSynchronize();
}

at::Tensor TestGridCUDA(
        const at::Tensor Points,
        const at::Tensor bbox_max,
        const at::Tensor bbox_min,
        int K,
        float r) {
    std::cout << "enter TestGrid" << std::endl;
    float r2 = r * r;
    float cell_size = r;
    GridParams params;
    int num_points = Points.size(0);
    SetupGridParamsCUDA(
        bbox_max.contiguous().data_ptr<float>(),
        bbox_min.contiguous().data_ptr<float>(),
        cell_size, params);
    // copy params to gpu
    // copy params to gpu
    GridParams* d_params;
    // std::cout << "d_params start" << std::endl;
    hipMalloc((void**)&d_params, sizeof(GridParams));
    // std::cout << "d_params allocated" << std::endl;
    hipMemcpy(d_params, &params, sizeof(GridParams), hipMemcpyHostToDevice);
    // std::cout << "d_params copied" << std::endl;
    // std::cout << params.gridMax.x << ' ' << params.gridMax.y << ' ' << params.gridMax.z << std::endl;
    // std::cout << "grid params setup done" << std::endl;

    auto int_dtype = Points.options().dtype(at::kInt);
    
    // not used right now
    at::Tensor Grid = at::full({params.gridTotal}, -1, int_dtype);
    // cell -> #points in this cell
    at::Tensor GridCnt = at::zeros({params.gridTotal}, int_dtype);
    // Point -> cell idx
    at::Tensor GridCell = at::full({num_points, 3}, -1, int_dtype);
    // Point -> next point idx in the same cell
    // at::Tensor GridNext = at::full({num_points}, -1, int_dtype);
    // Point -> idx in its cell
    at::Tensor GridIdx = at::full({num_points}, -1, int_dtype);

    InsertPointsCUDA(Points, Grid, GridCnt, GridCell, GridIdx, d_params);
    std::cout << "points inserted" << std::endl;

    return GridCnt;
}

at::Tensor PrefixSum(at::Tensor GridCnt) {
    int num_grids = GridCnt.size(0);
    at::Tensor GridOff = at::zeros({num_grids}, GridCnt.options());
    preallocBlockSumsInt(num_grids);
    prescanArrayRecursiveInt(
        GridOff.contiguous().data_ptr<int>(),
        GridCnt.contiguous().data_ptr<int>(),
        num_grids,
        0
    );
    hipDeviceSynchronize();
    deallocBlockSumsInt();
    return GridOff;
}

// fix K to be 5 for now
// template later
__global__ void FindNbrsGridCUDA(
    const float* __restrict__ Points, 
    const int* __restrict__ GridCnt,
    const int* __restrict__ GridCell,
    float* __restrict__ dists,
    float* __restrict__ idxs,
    GridParams& params,
    int num_points,
    float r2) {

    const int K = 5;

    int i = __mul24(blockIdx.x, blockDim.x) _ threadId.x;
    if (i >= num_points) return;

    float3 dist;
    float dsq;

    register float px = Points[i*3], py = Points[i*3+1], pz = Points[i*3+2];
    register float res_y = params.gridRes.y, res_z = params.gridRes.z;
    int cx = GridCell_a[i][0], cy = GridCell_a[i][1], cz = GridCell_a[i][2];
    int startx = std::max(0, cx-params.gridSrch), endx = std::min(cx+params.gridSrch, params.gridRes.x-1);
    int starty = std::max(0, cy-params.gridSrch), endy = std::min(cy+params.gridSrch, params.gridRes.y-1);
    int startz = std::max(0, cz-params.gridSrch), endz = std::min(cz+params.gridSrch, params.gridRes.z-1);

    float min_dists[5];
    int min_idxs[5];
    MinK<float, int> mink(min_dists, min_idxs, K)
    for (int x=startx; x<=endx; ++x) {
        for (int y=starty; y<=endy; ++y) {
            for (int z=startz; z<=endz; ++z) {
                int cur = (x*res_y + y)*res_z + z;
                int p_start = cur-1 >= 0 ? GridCnt[cur-1] : 0;
                int p_end = GridCnt[cur];

                for (int p=p_start; p < p_end; ++p) {
                    if (p != i || true) {
                        dist.x = Points[p*3] - px;
                        dist.y = Points[p*3+1] - py;
                        dist.z = Points[p*3+2] - pz;
                        dsq = dist.x*dist.x + dist.y*dist.y + dist.z*dist.z;
                        if (dsq <= r2) {
                            mink.add(dsq, p)
                        }
                    }
                }
                min.sort();
                for (int k=0; k < mink.size(); ++k) {
                    idxs[i*K+k] = min_idxs[k];
                    idxs[i*K+k] = min_dists[k];
                }
            }
        }
    }
}