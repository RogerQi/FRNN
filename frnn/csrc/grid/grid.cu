#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <tuple>

#include "grid.h"
#include "prefix_sum.h"
#include "counting_sort.h"
#include "utils/mink.cuh"
// customized dispatch utils for our function type
#include "utils/dispatch.h"


void SetupGridParams(
    float* bboxes,
    float cell_size,
    GridParams* params) {
  params->grid_min.x = bboxes[0];
  params->grid_max.x = bboxes[1];
  params->grid_min.y = bboxes[2];
  params->grid_max.y = bboxes[3];
  params->grid_min.z = bboxes[4];
  params->grid_max.z = bboxes[5];

  params->grid_size = params->grid_max - params->grid_min;
  float res_min = std::min(std::min(params->grid_size.x, params->grid_size.y), params->grid_size.z);
  if (cell_size < res_min/MAX_RES)
    cell_size = res_min / MAX_RES;
  params->grid_res.x = (int)(params->grid_size.x / cell_size) + 1;
  params->grid_res.y = (int)(params->grid_size.y / cell_size) + 1;
  params->grid_res.z = (int)(params->grid_size.z / cell_size) + 1;
  params->grid_total = params->grid_res.x * params->grid_res.y * params->grid_res.z;

  params->grid_delta = 1 / cell_size;

  return;
}

/*
void SetupGridParams(
    float* bboxes,
    float cell_size,
    float* params) {
  params[GRID_MIN_X] = bboxes[0];
  params[GRID_MIN_Y] = bboxes[2];
  params[GRID_MIN_Z] = bboxes[4];
  float grid_size_x = bboxes[1] - bboxes[0];
  float grid_size_y = bboxes[3] - bboxes[2];
  float grid_size_z = bboxes[5] - bboxes[4];
  float res_min = std::min(std::min(grid_size_x, grid_size_y), grid_size_z);
  if (cell_size < res_min/MAX_RES)
    cell_size = res_min / MAX_RES;
  params[GRID_RES_X] = (int)(grid_size_x / cell_size) + 1;
  params[GRID_RES_Y] = (int)(grid_size_y / cell_size) + 1;
  params[GRID_RES_Z] = (int)(grid_size_z / cell_size) + 1;
  params[GRID_TOTAL]= params[GRID_RES_X] * params[GRID_RES_Y] * params[GRID_RES_Z];
  params[GRID_DELTA] = 1 / cell_size;
  return;
}
*/

__global__ void InsertPointsKernel(
    const float* __restrict__ points,
    const long* __restrict__ lengths,
    const float* __restrict__ params,
    int* grid_cnt, // not sure if we can use __restrict__ here
    int* __restrict__ grid_cell,
    int* __restrict__ grid_idx,
    int N,
    int P,
    int G) {

  int chunks_per_cloud = (1 + (P - 1) / blockDim.x);
  int chunks_to_do = N * chunks_per_cloud;
  for (int chunk=blockIdx.x; chunk < chunks_to_do; chunk += gridDim.x) {
    int n = chunk / chunks_per_cloud;
    assert(n < N);
    int start_point = blockDim.x * (chunk % chunks_per_cloud);
    int p = start_point + threadIdx.x;
    if (p >= lengths[n])
      continue;
    assert(p < P);
    // if (p >= P)
    //   printf("p: %d; P: %d\n", p, P);

    float grid_min_x = params[n*GRID_PARAMS_SIZE+GRID_MIN_X];
    float grid_min_y = params[n*GRID_PARAMS_SIZE+GRID_MIN_Y];
    float grid_min_z = params[n*GRID_PARAMS_SIZE+GRID_MIN_Z];
    float grid_delta = params[n*GRID_PARAMS_SIZE+GRID_DELTA];
    int grid_res_x = params[n*GRID_PARAMS_SIZE+GRID_RES_X];
    int grid_res_y = params[n*GRID_PARAMS_SIZE+GRID_RES_Y];
    int grid_res_z = params[n*GRID_PARAMS_SIZE+GRID_RES_Z];

    int gc_x = (int) ((points[(n*P+p)*3+0]-grid_min_x) * grid_delta);
    int gc_y = (int) ((points[(n*P+p)*3+1]-grid_min_y) * grid_delta);
    int gc_z = (int) ((points[(n*P+p)*3+2]-grid_min_z) * grid_delta);

    gc_x = std::max(std::min(gc_x, grid_res_x-1), 0);
    gc_y = std::max(std::min(gc_y, grid_res_y-1), 0);
    gc_z = std::max(std::min(gc_z, grid_res_z-1), 0);

    int gs = (gc_x*grid_res_y + gc_y) * grid_res_z + gc_z;
    if (gs >= G)
      printf("gs: %d; G: %d;\n", gs, G);
    assert(gs < G);
    grid_cell[n*P+p] = gs;
    grid_idx[n*P+p] = atomicAdd(&grid_cnt[n*G + gs], 1);
  } 
}

void InsertPointsCUDA(
    const at::Tensor points,    // (N, P, 3)
    const at::Tensor lengths,   // (N,)
    const at::Tensor params,    // (N, 8)
    at::Tensor grid_cnt,        // (N, G)
    at::Tensor grid_cell,       // (N, P)      
    at::Tensor grid_idx,        // (N, P)
    int G) {
  
  at::TensorArg points_t{points, "points", 1};
  at::TensorArg lengths_t{lengths, "lengths", 2};
  at::TensorArg params_t{params, "params", 3};
  at::TensorArg grid_cnt_t{grid_cnt, "grid_cnt", 4};
  at::TensorArg grid_cell_t{grid_cell, "grid_cell", 5};
  at::TensorArg grid_idx_t{grid_idx, "grid_idx", 6};

  at::CheckedFrom c = "InsertPointsCUDA";
  at::checkAllSameGPU(c, {points_t, lengths_t, params_t, grid_cnt_t, grid_cell_t, grid_idx_t});
  at::checkAllSameType(c, {grid_cnt_t, grid_cell_t, grid_idx_t});

  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int threads = 256;
  int blocks = 256;

  InsertPointsKernel<<<blocks, threads, 0, stream>>>(
    points.contiguous().data_ptr<float>(),
    lengths.contiguous().data_ptr<long>(),
    params.contiguous().data_ptr<float>(),
    grid_cnt.contiguous().data_ptr<int>(),
    grid_cell.contiguous().data_ptr<int>(),
    grid_idx.contiguous().data_ptr<int>(),
    points.size(0),
    points.size(1),
    G
  );
  AT_CUDA_CHECK(hipGetLastError());
}


template<int K>
__global__ void FindNbrsKernel(
    const float* __restrict__ points1,       
    const float* __restrict__ points2,       
    const long* __restrict__ lengths1,        
    const long* __restrict__ lengths2,
    const int* __restrict__ grid_off,
    const int* __restrict__ sorted_points1_idxs,
    const int* __restrict__ sorted_points2_idxs,
    const float* __restrict__ params,
    float* __restrict__ dists,               
    long* __restrict__ idxs,                  
    int N,
    int P1,
    int P2,
    int G,
    float r) {
  float min_dists[K];
  int min_idxs[K];
  float3 diff;
  float sqdist;
  float r2 = r*r;
  
  int chunks_per_cloud = (1 + (P1 - 1) / blockDim.x);
  int chunks_to_do = N * chunks_per_cloud;
  for (int chunk = blockIdx.x; chunk < chunks_to_do; chunk += gridDim.x) {
    int n = chunk / chunks_per_cloud;
    int start_point = blockDim.x * (chunk % chunks_per_cloud);
    int p1 = start_point + threadIdx.x;
    if (p1 >= lengths1[n])
      continue;
    float3 cur_point;
    cur_point.x = points1[n*P1*3 + p1*3];
    cur_point.y = points1[n*P1*3 + p1*3 + 1];
    cur_point.z = points1[n*P1*3 + p1*3 + 2];

    float grid_min_x = params[n*GRID_PARAMS_SIZE+GRID_MIN_X];
    float grid_min_y = params[n*GRID_PARAMS_SIZE+GRID_MIN_Y];
    float grid_min_z = params[n*GRID_PARAMS_SIZE+GRID_MIN_Z];
    float grid_delta = params[n*GRID_PARAMS_SIZE+GRID_DELTA];
    int grid_res_x = params[n*GRID_PARAMS_SIZE+GRID_RES_X];
    int grid_res_y = params[n*GRID_PARAMS_SIZE+GRID_RES_Y];
    int grid_res_z = params[n*GRID_PARAMS_SIZE+GRID_RES_Z];
    int grid_total = params[n*GRID_PARAMS_SIZE+GRID_TOTAL];

    int min_gc_x = (int) std::floor((cur_point.x-grid_min_x-r) * grid_delta);
    int min_gc_y = (int) std::floor((cur_point.y-grid_min_y-r) * grid_delta);
    int min_gc_z = (int) std::floor((cur_point.z-grid_min_z-r) * grid_delta);
    int max_gc_x = (int) std::floor((cur_point.x-grid_min_x+r) * grid_delta);
    int max_gc_y = (int) std::floor((cur_point.y-grid_min_y+r) * grid_delta);
    int max_gc_z = (int) std::floor((cur_point.z-grid_min_z+r) * grid_delta);
    MinK<float, int> mink(min_dists, min_idxs, K);
    for (int x=std::max(min_gc_x, 0); x<=std::min(max_gc_x, grid_res_x-1); ++x) {
      for (int y=std::max(min_gc_y, 0); y<=std::min(max_gc_y, grid_res_y-1); ++y) {
        for (int z=std::max(min_gc_z, 0); z<=std::min(max_gc_z, grid_res_z-1); ++z) {
          int cell_idx = (x*grid_res_y + y)*grid_res_z + z;
          int p2_start = grid_off[n*G + cell_idx];
          int p2_end;
          if (cell_idx+1 == grid_total) {
            p2_end = lengths2[n];
          }
          else {
            p2_end = grid_off[n*G+cell_idx+1]; 
          }
          for (int p2=p2_start; p2<p2_end; ++p2) {
            diff.x = points2[n*P2*3 + p2*3] - cur_point.x;
            diff.y = points2[n*P2*3 + p2*3 + 1] - cur_point.y;
            diff.z = points2[n*P2*3 + p2*3 + 2] - cur_point.z;
            sqdist = diff.x*diff.x + diff.y*diff.y + diff.z*diff.z;
            if (sqdist <= r2) {
              mink.add(sqdist, sorted_points2_idxs[n*P2+p2]);
            }
          }
        }
      }
    }
    // TODO: add return_sort here
    mink.sort();
    int old_p1 = sorted_points1_idxs[p1];
    for (int k=0; k < mink.size(); ++k) {
      idxs[n*P1*K + old_p1*K + k] = min_idxs[k];
      dists[n*P1*K + old_p1*K + k] = min_dists[k];
    }
  }
}

template<int K>
struct FindNbrsKernelFunctor {
  static void run(
      size_t blocks,
      size_t threads,
      const float* __restrict__ points1,          // (N, P1, 3)
      const float* __restrict__ points2,          // (N, P2, 3)
      const long* __restrict__ lengths1,          // (N,)
      const long* __restrict__ lengths2,          // (N,)
      const int* __restrict__ pc2_grid_off,           // (N, G)
      const int* __restrict__ sorted_points1_idxs,   // (N, P)
      const int* __restrict__ sorted_points2_idxs,   // (N, P)
      const float* __restrict__ params,           // (N,)
      float* __restrict__ dists,                  // (N, P1, K)
      long* __restrict__ idxs,                    // (N, P1, K)
      int N,
      int P1,
      int P2,
      int G,
      float r) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    FindNbrsKernel<K><<<blocks, threads, 0, stream>>>(
      points1, points2, lengths1, lengths2, pc2_grid_off, 
      sorted_points1_idxs, sorted_points2_idxs, params,
      dists, idxs, N, P1, P2, G, r);
  }
};

// TODO: figure out max & min; sanity check in python
constexpr int MIN_K = 1;
constexpr int MAX_K = 32;

std::tuple<at::Tensor, at::Tensor> FindNbrsCUDA(
    const at::Tensor points1,
    const at::Tensor points2,
    const at::Tensor lengths1,
    const at::Tensor lengths2,
    const at::Tensor pc2_grid_off,
    const at::Tensor sorted_points1_idxs,
    const at::Tensor sorted_points2_idxs,
    const at::Tensor params,
    int K,
    float r) {
  at::TensorArg points1_t{points1, "points1", 1};
  at::TensorArg points2_t{points2, "points2", 2};
  at::TensorArg lengths1_t{lengths1, "lengths1", 3};
  at::TensorArg lengths2_t{lengths2, "lengths2", 4};
  at::TensorArg pc2_grid_off_t{pc2_grid_off, "pc2_grid_off", 5};
  at::TensorArg sorted_points1_idxs_t{sorted_points1_idxs, "sorted_points1_idxs", 6};
  at::TensorArg sorted_points2_idxs_t{sorted_points2_idxs, "sorted_points2_idxs", 7};
  at::TensorArg params_t{params, "params", 8};

  at::CheckedFrom c = "FindNbrsCUDA";
  at::checkAllSameGPU(c, {points1_t, points2_t, lengths1_t, lengths2_t, pc2_grid_off_t, sorted_points1_idxs_t, sorted_points2_idxs_t, params_t});
  at::checkAllSameType(c, {points1_t, points2_t});
  at::checkAllSameType(c, {lengths1_t, lengths2_t});
  at::checkAllSameType(c, {pc2_grid_off_t, sorted_points1_idxs_t, sorted_points2_idxs_t});
  at::cuda::CUDAGuard device_guard(points1.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int N = points1.size(0);
  int P1 = points1.size(1);
  int P2 = points2.size(1);
  int G = pc2_grid_off.size(1);
  
  auto idxs = at::full({N, P1, K}, -1, lengths1.options());
  auto dists = at::full({N, P1, K}, -1, points1.options());

  int threads = 256;
  int blocks = 256;

  DispatchKernel1D<FindNbrsKernelFunctor, MIN_K, MAX_K>( 
    K,
    blocks,
    threads,
    points1.contiguous().data_ptr<float>(),
    points2.contiguous().data_ptr<float>(),
    lengths1.contiguous().data_ptr<long>(),
    lengths2.contiguous().data_ptr<long>(),
    pc2_grid_off.contiguous().data_ptr<int>(),
    sorted_points1_idxs.contiguous().data_ptr<int>(),
    sorted_points2_idxs.contiguous().data_ptr<int>(),
    params.contiguous().data_ptr<float>(),
    dists.data_ptr<float>(),
    idxs.data_ptr<long>(),
    N,
    P1,
    P2,
    G,
    r
  );

  return std::make_tuple(idxs, dists);
}