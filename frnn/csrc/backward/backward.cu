#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <float.h>
#include <iostream>
#include <tuple>

__global__ void FRNNBackwardKernel(
    const float* __restrict__ points1,
    const float* __restrict__ points2,
    const long* __restrict__ lengths1,
    const long* __restrict__ lengths2,
    const long* __restrict__ idxs,
    const float* __restrict__ grad_dists,
    float* __restrict__ grad_points1,
    float* __restrict__ grad_points2,
    int N,
    int P1,
    int P2,
    int K) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = gridDim.x * blockDim.x;
  const int D = 3; // only support 3D tensor now
  for (int i = tid; i < N * P1 * K * D; i += stride) {
    const int n = i / (P1 * K * D);
    int rem = i % (P1 * K * D);
    const int p1_idx = rem / (K * D);
    rem = rem % (K * D);
    const int k = rem / D;
    const int d = rem % D;

    const long num1 = lengths1[n];
    const long num2 = lengths2[n];
    if ((p1_idx < num1) && (k < num2)) {
      const long p2_idx = idxs[n * P1 * K + p1_idx * K + k];
      if (p2_idx < 0) // sentinel value -1 indicating no fixed radius negihbors here
        continue;
      const float grad_dist = grad_dists[n * P1 * K + p1_idx * K + k];

      const float diff = 2.0f * grad_dist * 
          (points1[n * P1 * D + p1_idx * D + d] - points2[n * P2 * D + p2_idx * D + d]);
      atomicAdd(grad_points1 + n * P1 * D + p1_idx * D + d, diff);
      atomicAdd(grad_points2 + n * P2 * D + p2_idx * D + d, -1.0f * diff);
    }
  }
}


std::tuple<at::Tensor, at::Tensor> FRNNBackwardCUDA(
    const at::Tensor points1,
    const at::Tensor points2,
    const at::Tensor lengths1,
    const at::Tensor lengths2,
    const at::Tensor idxs,
    const at::Tensor grad_dists) {
  
  at::TensorArg points1_t{points1, "points1", 1}, points2_t{points2, "points2", 2},
      lengths1_t{lengths1, "lenghts1", 3}, lengths2_t{lengths2, "lengths2", 4},
      idxs_t{idxs, "idxs", 5}, grad_dists_t{grad_dists, "grad_dists", 6};
  at::CheckedFrom c = "FRNNBackwardCUDA";
  at::checkAllSameGPU(c, {points1_t, points2_t, lengths1_t, lengths2_t, idxs_t, grad_dists_t});
  at::checkAllSameType(c, {points1_t, points2_t, grad_dists_t});
  at::checkAllSameType(c, {lengths1_t, lengths2_t, idxs_t});

  const int N = points1.size(0);
  const int P1 = points1.size(1);
  const int P2 = points2.size(1);
  const int K = idxs.size(2);
  const int D = 3;

  TORCH_CHECK(points1.size(2) == 3 && points2.size(2) == 3, "Only 3D points are supported");
  TORCH_CHECK(idxs.size(0) == N, "FRNN idxs must have the same batch dimension");
  TORCH_CHECK(idxs.size(1) == P1, "FRNN idxs must have the same point dimension as P1");
  TORCH_CHECK(grad_dists.size(0) == N);
  TORCH_CHECK(grad_dists.size(1) == P1);
  TORCH_CHECK(grad_dists.size(2) == K);

  at::Tensor grad_points1 = at::zeros({N, P1, D}, points1.options());
  at::Tensor grad_points2 = at::zeros({N, P2, D}, points2.options());

  if (grad_points1.numel() == 0 || grad_points2.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(grad_points1, grad_points2);
  }

  const int blocks = 64;
  const int threads = 512;
  
  at::cuda::CUDAGuard device_guard(points1.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  FRNNBackwardKernel<<<blocks, threads, 0, stream>>>(
    points1.contiguous().data_ptr<float>(),
    points2.contiguous().data_ptr<float>(),
    lengths1.contiguous().data_ptr<long>(),
    lengths2.contiguous().data_ptr<long>(),
    idxs.contiguous().data_ptr<long>(),
    grad_dists.contiguous().data_ptr<float>(),
    grad_points1.data_ptr<float>(),
    grad_points2.data_ptr<float>(),
    N,
    P1,
    P2,
    K);
    
  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(grad_points1, grad_points2);
}